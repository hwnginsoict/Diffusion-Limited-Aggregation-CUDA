#include <iostream>
#include <sys/time.h>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

__device__ int width;
__device__ int height;
__device__ int* map;

int cwidth;
int cheight;
int* cmap;

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

typedef struct {
    int x;
    int y;
    int state;
    uint seed;
} Point;

__device__ float random_number(int seed){
    int x = seed;
    int m = 65537;
    int a = 75;
    int k = 10;
    for(int i=0; i<k; i++){
      x = (a*x)%m;
    }
    float ans = (float)x/(float)m;
    return ans;
}

__global__ void init_particles(Point* points, int number) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number) {
        Point p;
        p.state = 0;
        p.x = (int)(random_number(i)*width);
        p.y = (int)(random_number((int)(random_number(i)*65537))*height);
        p.seed = i;
        points[i] = p;
    }
}

__device__ void move(Point& p) {
    p.seed = (int)(random_number(p.seed)*65537);
    int mov = p.seed % 4;
    if (mov == 0) {
        p.x += 1;
    } else if (mov == 1) {
        p.x -= 1;
    } else if (mov == 2) {
        p.y += 1;
    } else {
        p.y -= 1;
    }

    // condition
    if (p.x >= width) {
        p.x = 0;
    } else if (p.x < 0) {
        p.x = width - 1;
    }

    if (p.y >= height) {
        p.y = 0;
    } else if (p.y < 0) {
        p.y = height - 1;
    }
}

__device__ int check_occupied(int x, int y) {
    for (int x1 = -1; x1 <= 1; x1++) {
        for (int y1 = -1; y1 <= 1; y1++) {
            if (y1 == 0 && x1 == 0) continue;
            if (x1 + x < 0 || x1 + x >= width) continue;
            if (y1 + y < 0 || y1 + y >= height) continue;
            int j = ((y + y1) * width) + (x + x1);
            if (map[j] > 0) {
                return 1;
            }
        }
    }
    return 0;
}

__device__ void occupy(Point& p) {
    p.state = 1;
    int i = p.y * width + p.x;
    map[i] = 1;
}

__global__ void fill(Point* points, int number, int* changed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number) {
        Point p = points[i];
        if (p.state == 1) return;
        if (check_occupied(p.x, p.y)) {
            points[i].state = 1;
            int idx = p.y * width + p.x;
            map[idx] = 1;
            *changed = 1; // Indicate that a change was made
        }
    }
}

__global__ void move_particles(Point* points, int number) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number) {
        if (points[i].state == 0) {
            move(points[i]);
        }
    }
}

void save_map(const char* file_name, int* map, int width, int height) {
    FILE* fp = fopen(file_name, "w");
    if (fp == NULL) {
        printf("Error opening the file %s\n", file_name);
        return;
    }
    for (int i = 0; i < width * height; i++) {
        if (i % width == 0) {
            fprintf(fp, "\n");
        }
        fprintf(fp, "%d ", map[i]);
    }
    fclose(fp);
}

int main() {
    cwidth = 200;
    cheight = 200;
    int number = 3000;
    int steps = 10000;
    int sx = 100;
    int sy = 100;
    int si = sy * cwidth + sx;

    double iStart = cpuSecond();

    cmap = (int*)malloc(sizeof(int) * cwidth * cheight);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &cwidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(height), &cheight, sizeof(int));
    int* d_map;
    hipMalloc(&d_map, sizeof(int) * cwidth * cheight);
    hipMemcpyToSymbol(HIP_SYMBOL(map), &d_map, sizeof(int*));
    hipMemcpy(d_map, cmap, sizeof(int) * cwidth * cheight, hipMemcpyHostToDevice);

    Point* points;
    hipMalloc(&points, sizeof(Point) * number);

    const int base = 1;
    int init_threads = base;
    int init_blocks = (number + base - 1) / base;
    if (number < base) {
        init_threads = number;
        init_blocks = 1;
    }
    init_particles<<<init_blocks, init_threads>>>(points, number);
    hipDeviceSynchronize();

    printf("Initialization time: %.3f milliseconds\n", 1000 * (cpuSecond() - iStart));
    double sStart = cpuSecond();

    // Initialize starting point on device
    int initial_value = 1;
    hipMemcpy(&d_map[si], &initial_value, sizeof(int), hipMemcpyHostToDevice);

    int* d_changed;
    hipMalloc(&d_changed, sizeof(int));

    for (int i = 0; i < steps; i++) {
        int changed;
        do {
            changed = 0;
            hipMemcpy(d_changed, &changed, sizeof(int), hipMemcpyHostToDevice);

            fill<<<init_blocks, init_threads>>>(points, number, d_changed);
            hipDeviceSynchronize();

            hipMemcpy(&changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        } while (changed);

        move_particles<<<init_blocks, init_threads>>>(points, number);
        hipDeviceSynchronize();
    }

    printf("Simulation time: %.3f milliseconds\n", 1000 * (cpuSecond() - sStart));

    hipMemcpy(cmap, d_map, sizeof(int) * cwidth * cheight, hipMemcpyDeviceToHost);

    save_map("output_map.txt", cmap, cwidth, cheight);
    free(cmap);
    hipFree(d_map);
    hipFree(points);
    hipFree(d_changed);
    return 0;
}